/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"

#define BLOCK_WIDTH 32

using namespace std;


// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

/*
// Allocate a 2D array
double **alloc2D(int m,int n){
   double **E;
   int nx=n, ny=m;
   E = (double**)malloc(sizeof(double*)*ny + sizeof(double)*nx*ny);
   assert(E);
   int j;
   for(j=0;j<ny;j++) 
     E[j] = (double*)(E+ny) + j*nx;
   return(E);
}
*/
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
 double stats(double *E, int m, int n, double *_mx){
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j=1; j<=m; j++)
       for (i=1; i<=n; i++) {
       l2norm += E[j*(n+2)+i]*E[j*(n+2)+i];
       if (E[j*(n+2)+i] > mx)
           mx = E[j*(n+2)+i];
      }
     *_mx = mx;
     l2norm /= (double) ((m)*(n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], float& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__global__ void simulateKernal(double* E, double* E_prev, double* R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b){

    //int tile_d=(WIDTH+2)*(WIDTH+2);

    __shared__ double tile_E_prev[34*34];

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((col==0)&&(row<m)) E_prev[(row+1)*(n+2)]=E_prev[(row+1)*(n+2)+2];
    if((col==(n-1))&&(row<m)) E_prev[(row+1)*(n+2)+n+1]=E_prev[(row+1)*(n+2)+n-1];
    if((row==0)&&(col<n)) E_prev[col+1]=E_prev[2*(n+2)+col+1];
    if((row==(m-1))&&(col<n)) E_prev[(m+1)*(n+2)+col+1]=E_prev[(m-1)*(n+2)+col+1];
    __syncthreads();


    int tile_x=threadIdx.y+1;
    int tile_y=threadIdx.x+1;

    if((row<m)&&(col<n)) tile_E_prev[tile_x*(blockDim.x+2)+tile_y]=E_prev[(row+1)*(n+2)+col+1];
    if(tile_x==1) tile_E_prev[tile_y]=E_prev[row*(n+2)+col+1];
    if(tile_x==blockDim.y) tile_E_prev[(blockDim.y+1)*(blockDim.x+2)+tile_y]=E_prev[(row+2)*(n+2)+col+1];
    if(tile_y==1) tile_E_prev[tile_x*(blockDim.x+2)]=E_prev[(row+1)*(n+2)+col];
    if(tile_y==blockDim.x) tile_E_prev[tile_x*(blockDim.x+2)+(blockDim.x+1)]=E_prev[(row+1)*(n+2)+col+2];

    __syncthreads();

    if((row<m)&&(col<n)){

        E[(row+1)*(n+2)+(col+1)] = tile_E_prev[tile_x*(blockDim.x+2)+tile_y]+alpha*(tile_E_prev[tile_x*(blockDim.x+2)+tile_y+1]+tile_E_prev[tile_x*(blockDim.x+2)+tile_y-1]-4*tile_E_prev[tile_x*(blockDim.x+2)+tile_y]+tile_E_prev[(tile_x+1)*(blockDim.x+2)+tile_y]+tile_E_prev[(tile_x-1)*(blockDim.x+2)+tile_y]);

        double e=E[(row+1)*(n+2)+col+1];
        double r=R[(row+1)*(n+2)+col+1];


        e = e-dt*(kk* e*(e- a)*(e-1)+ e *r);

        r = r + dt*(epsilon+M1* r/(e+M2))*(-r-kk* e*(e-b-1));

        E[(row+1)*(n+2)+col+1]=e;
        R[(row+1)*(n+2)+col+1]=r;

    }

    __syncthreads();

    if((row<m)&&(col<n)){
        double tmp = E[(row+1)*(n+2)+col+1];
        E[(row+1)*(n+2)+col+1] = E_prev[(row+1)*(n+2)+col+1];
        E_prev[(row+1)*(n+2)+col+1] = tmp;
    }
    __syncthreads();

}

/*
void simulate (double* E,  double* E_prev,double* R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b)
{
  int i, j; 
    /* 
     * Copy data from boundary of the computational box 
     * to the padding region, set up for differencing
     * on the boundary of the computational box
     * Using mirror boundaries
     */
/*

    for (j=1; j<=m; j++) 
      E_prev[j*(m+2)] = E_prev[j*(m+2)+2];
    for (j=1; j<=m; j++) 
      E_prev[j*(m+2)+n+1] = E_prev[j*(m+2)+n-1];
    
    for (i=1; i<=n; i++) 
      E_prev[i] = E_prev[2*(m+2)+i];
    for (i=1; i<=n; i++) 
      E_prev[(m+1)*(m+2)+i] = E_prev[(m-1)*(m+2)+i];

    
    // Solve for the excitation, the PDE
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++) {
    E[j*(m+2)+i] = E_prev[j*(m+2)+i]+alpha*(E_prev[j*(m+2)+i+1]+E_prev[j*(m+2)+i-1]-4*E_prev[j*(m+2)+i]+E_prev[(j+1)*(m+2)+i]+E_prev[(j-1)*(m+2)+i]);
      }
    }
    
    /* 
     * Solve the ODE, advancing excitation and recovery to the
     *     next timtestep
     */

/*
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
    E[j*(m+2)+i] = E[j*(m+2)+i] -dt*(kk* E[j*(m+2)+i]*(E[j*(m+2)+i] - a)*(E[j*(m+2)+i]-1)+ E[j*(m+2)+i] *R[j*(m+2)+i]);
    }
    
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
    R[j*(m+2)+i] = R[j*(m+2)+i] + dt*(epsilon+M1* R[j*(m+2)+i]/( E[j*(m+2)+i]+M2))*(-R[j*(m+2)+i]-kk* E[j*(m+2)+i]*(E[j*(m+2)+i]-b-1));
    }
    
}

*/

// Main program
int main (int argc, char** argv)
{
  /*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
  double *E, *R, *E_prev;
  
  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;
  
  float T=1000.0;
  int m=200,n=200;
  int plot_freq = 0;
  int px = 1, py = 1;
  int no_comm = 0;
  int num_threads=1; 

  cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
  m = n;  
  int size=(m+2)*(n+2);

  // Allocate contiguous memory for solution arrays
  // The computational box is defined on [1:m+1,1:n+1]
  // We pad the arrays in order to facilitate differencing on the 
  // boundaries of the computation box

  E=(double *)malloc(size*sizeof(double));
  E_prev=(double *)malloc(size*sizeof(double));
  R=(double *)malloc(size*sizeof(double));
  
  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++)
      E_prev[j*(n+2)+i] = R[j*(n+2)+i] = 0;
  
  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j*(n+2)+i] = 1.0;
  
  for (j=m/2+1; j<=m; j++)
    for (i=1; i<=n; i++)
      R[j*(n+2)+i] = 1.0;
  
  double dx = 1.0/n;

  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  cout << "Grid Size       : " << n << endl; 
  cout << "Duration of Sim : " << T << endl; 
  cout << "Time step dt    : " << dt << endl; 
  cout << "Process geometry: " << px << " x " << py << endl;
  if (no_comm)
    cout << "Communication   : DISABLED" << endl;
  
  cout << endl;
  
  // Start the timer
  double t0 = getTime();
  
 
  // Simulated time is different from the integer timestep number
  // Simulated time
  double t = 0.0;
  // Integer timestep number
  int niter=0;

  double *d_E, *d_E_prev, *d_R;

  hipMalloc((void **) &d_E, size*sizeof(double));
  hipMemcpy(d_E, E, size*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **) &d_E_prev, size*sizeof(double));
  hipMemcpy(d_E_prev, E_prev, size*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **) &d_R, size*sizeof(double));
  hipMemcpy(d_R, R, size*sizeof(double), hipMemcpyHostToDevice);

  int tnx=n/BLOCK_WIDTH;
  int tny=m/BLOCK_WIDTH;

  if(n%BLOCK_WIDTH) tnx++;
  if(m%BLOCK_WIDTH) tny++;

  dim3 dimGrid(tnx,tny,1);
  dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH,1);

  double t1 = getTime();
  
  while (t<T) {
    
    t += dt;
    niter++;

    simulateKernal<<<dimGrid,dimBlock>>>(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
    hipDeviceSynchronize();

    //simulate(E, E_prev, R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
    
    //swap current E with previous E
    //double *tmp = E; E = E_prev; E_prev = tmp;
    
    if (plot_freq){

      int k = (int)(t/plot_freq);
      if ((t - k * plot_freq) < dt){
        hipMemcpy(E, d_E, size*sizeof(double), hipMemcpyDeviceToHost);
	splot(E,t,niter,m+2,n+2);
      }
    }
  }//end of while loop

  double time_elapsed1 = getTime() - t1;

  hipMemcpy(E, d_E, size*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(E_prev, d_E_prev, size*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(R, d_R, size*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_E);
  hipFree(d_E_prev);
  hipFree(d_R);

  double time_elapsed = getTime() - t0;

  double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed ;
  double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;
  cout << "BLOCK SIZE                  : " << BLOCK_WIDTH << endl;
  cout << "Number of Iterations        : " << niter << endl;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
  cout << "Time (no data transfer)     : " << time_elapsed1 <<endl;
  cout << "Sustained Gflops Rate       : " << Gflops << endl; 
  cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl;
  cout << "************************************************************" <<endl;
  cout << "************************************************************" <<endl;
  cout << "************************************************************" <<endl;

  //for(i=1;i<=m;i++) printf("E[%d,%d]:%f\n",i,i,E_prev[i*(m+2)+i]);

  double mx;
  double l2norm = stats(E_prev,m,n,&mx);
  cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

  if (plot_freq){
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }
  
  free (E);
  free (E_prev);
  free (R);
  
  return 0;
}
